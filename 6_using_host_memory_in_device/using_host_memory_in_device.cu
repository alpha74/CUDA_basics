#include "hip/hip_runtime.h"
// Using host memory from device kernel
// Author: alpha74

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>

using namespace std;

const int N = 100;

__global__ void add_next(int *a, int *r)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid < N)
	{
		if (tid == 0)
			r[tid] = 0;
		else
			r[tid] = a[tid] + a[tid - 1];	// Staggered addition
	}
}

int main()
{
	int *a, *result;
	int *dev_a, *dev_result;

	// Allocate memory on host
	hipHostAlloc((void**)&a, N * sizeof( int ) ,hipHostMallocDefault );
	hipHostAlloc((void**)&result, N * sizeof( int ), hipHostMallocDefault);

	// Get corresponding Device pointers
	hipHostGetDevicePointer(&dev_a, a, 0);
	hipHostGetDevicePointer(&dev_result, result, 0);

	// Initialize the array
	for (int i = 0; i < N; i++)
	{
		a[i] = 1 ;
	}

	// Launch kernel
	add_next << <4, 25 >> > (dev_a, dev_result);

	// Printing result
	hipDeviceSynchronize();

	cout << "\n\n Result: ";
	for (int i = 0; i < N ; i++)
	{
		cout << "\n " << result[i];
	}

	// Freeing memory
	hipHostFree(a);
	hipHostFree(result);
}
