#include "hip/hip_runtime.h"
// Demonstrate how thread are executed
// Author: alpha74

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void kernel()
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;

	int a;
	printf("\n [ %d ] Declared a.", tid);

	int b;
	printf("\n [ %d ] Declared b.", tid);

	a = 9;
	printf("\n [ %d ] Initialized a.", tid);

	b = 8;
	printf("\n [ %d ] Initialized b.", tid);
}

int main()
{
	// Launch kernel
	kernel << <1, 5 >> > ();
}
