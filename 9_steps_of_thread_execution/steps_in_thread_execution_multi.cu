#include "hip/hip_runtime.h"
// Demonstrates the steps in thread execution. Here less threads are launched than needed.
// Author: alpha74

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void kernel()
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;

	while (tid < 20)  // 5 threads are launched and 20 jobs are to be done.
	{

		int a;
		printf("\n [ %d ] Declared a.", tid);

		int b;
		printf("\n [ %d ] Declared b.", tid);

		a = 9;
		printf("\n [ %d ] Initialized a.", tid);

		b = 8;
		printf("\n [ %d ] Initialized b.", tid);


		tid += gridDim.x * blockDim.x;
	}
}

int main()
{
	// Launch kernel
	kernel << <1, 5 >> > ();
}
