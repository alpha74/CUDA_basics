// Basic CUDA Program
// Author: alpha74

#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void kernel();
int main()
{
	printf("\n Hello HOST ");

	kernel <<<1, 1>>> ();

	
	printf("\n");
	return 0;
}

__global__ void kernel()
{
	printf("\n Hello KERNEL" );
}
