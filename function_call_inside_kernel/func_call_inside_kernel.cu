#include "hip/hip_runtime.h"
// Using function calls inside kernel
// Author: alpha74

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>

using namespace std;

__device__ int fact(int n )
{
	int res = 1;

	for (int i = n; i > 1; i--)
	{
		res = res * i;
	}
	return res;
}

#define size 100

// Calculate factorial of numbers in an array. Using a fact() function.
__global__ void array_fact(int *arr )
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;

	while (tid < size)
	{
		// Decreasing the number to prevent numbers greater than range.
		arr[tid] = fact(tid/20 + 1 );
		
		tid += gridDim.x * blockDim.x ;
	}

}

int main()
{
	int arr[size];
	int *dev_arr;

	// Allocate array
	hipMalloc((void **)&dev_arr, size * sizeof(int));

	// Launch kernel
	array_fact << <5, 20 >> > (dev_arr );

	// Wait for GPU execution
	hipDeviceSynchronize();

	// Copy and print contents
	hipMemcpy(arr, dev_arr, size * sizeof(int), hipMemcpyDeviceToHost);

	cout << "\n factorial array: ";
	for (int i = 0; i < size; i++)
	{
		cout << " " << arr[i];
	}

	// Free memory
	hipFree(dev_arr);
}
