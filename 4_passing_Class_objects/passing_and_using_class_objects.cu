#include "hip/hip_runtime.h"

// Testing class objects passing

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>

using namespace std;

class Marks
{
	public:
		int m1;
		int m2;

		// Default ctor
		Marks()
		{
			m1 = 0;
			m2 = 0;
		}
};

const int N = 10;

__global__ void add(Marks *A, int * sum)
{
	int tid = blockIdx.x;

	printf("\n blockIdx.x = %d", tid);

	if (tid < N)
	{
		printf("\n tid: %d", tid);
		sum[tid] = A[tid].m1 + A[tid].m2;
	}
}

int main()
{
	Marks A[ N ];
	Marks *dev_A;

	int sum[N];
	int *dev_sum;

	// Device memory
	hipMalloc((void**)&dev_A, N * sizeof(Marks));
	hipMalloc((void**)&dev_sum, N * sizeof(int));

	// Setting marks
	for (int i = 0; i < N; i++)
	{
		A[i].m1 = 9;
		A[i].m2 = 8;
	}

	hipMemcpy(dev_A, A, N * sizeof(Marks), hipMemcpyHostToDevice);
	add << <N, 1 >> > (dev_A, dev_sum);

	hipMemcpy(sum, dev_sum, N * sizeof(int), hipMemcpyDeviceToHost);

	cout << "\n Sums: \n ";
	for (int i = 0; i < N; i++)
		cout << "\n " << i + 1 << ": " << sum[i];
}
