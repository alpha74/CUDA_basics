// Passing array of a Class and assigning elements at odd/even elements to another array.
// @alpha74

#include "hip/hip_runtime.h"


#include <iostream>
#include "stdio.h"

using namespace std;

class Coord
{
	int x;
	int y;

	public:
		Coord() 
		{
			x = 0;
			y = 0;
		}
		void set(int a, int b)
		{
			x = a;
			y = b;
		}
		void print()
		{
			printf(" (%d,%d) ", x, y);
		}
};

__global__ void foo( int lim, Coord C[], Coord *oddi, Coord *eveni)
{
	int tid = blockIdx.x;

	if (tid < lim)
	{
		if (tid % 2 == 0)
		{
			eveni[tid / 2] = C[tid];
		}
		else
		{
			oddi[tid / 2] = C[tid];
		}
	}
}

int main()
{
	const int N = 20;

	// Declare an array of size N
	Coord C[N];
	Coord result_odd[ N/2], result_even[ N/2 ];

	// Initialize the elements
	for (int i = 0; i < N; i++)
	{
		if (i % 2 == 0)
		{
			C[i].set(2, 2);
		}
		else
			C[i].set(-1, -1);
	}

	// Declare and allocate device memory
	Coord *dev_C;
	Coord *dev_odd, *dev_even;

	hipMalloc((void**)&dev_C, N * sizeof(Coord));
	hipMalloc((void**)&dev_odd, N / 2 * sizeof(Coord));
	hipMalloc((void**)&dev_even, N / 2 * sizeof(Coord));

	hipMemcpy(dev_C, C, N * sizeof(Coord), hipMemcpyHostToDevice);
	hipMemcpy(dev_odd, C, N/2 * sizeof(Coord), hipMemcpyHostToDevice);	// Adding part of the array for comparison.
	hipMemcpy(dev_even, C, N / 2 * sizeof(Coord), hipMemcpyHostToDevice);	// Adding part of the array for comparison.

	foo <<<N, 1 >>> ( N, dev_C, dev_odd, dev_even );

	// Copying back the results
	hipMemcpy(&result_even, dev_even, N/2 * sizeof(Coord), hipMemcpyDeviceToHost);
	hipMemcpy(&result_odd, dev_odd, N/2 * sizeof(Coord), hipMemcpyDeviceToHost);

	cout << "\n At even pos: ";
	for (int i = 0 ; i < N / 2; i++)
	{
		result_even[i].print();
	}

	cout << "\n At odd pos: ";
	for (int i = 0; i < N / 2; i++)
	{
		result_odd[i].print();
	}

	cout << "\n ";

	// Freeing device memory

	hipFree(dev_C);
	hipFree(dev_odd);
	hipFree(dev_even);

	return 0;
}
