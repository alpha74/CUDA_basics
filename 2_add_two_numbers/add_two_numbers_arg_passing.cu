#include "hip/hip_runtime.h"
// Passing Arguments to Kernel and Adding two numbers
// Author: alpha74

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void add(int, int, int *);	// Add two numbers

int main()
{
	int c;
	int *dev_c;

	hipMalloc((void**)&dev_c, sizeof(int));

	add << <1, 1 >> > (23, 10, dev_c);

	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("Result: %d", c);
	hipFree(dev_c);
}

__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}
