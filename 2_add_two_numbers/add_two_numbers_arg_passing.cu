#include "hip/hip_runtime.h"
// Passing Arguments to Kernel and Adding two numbers
// Author: alpha74

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void add(int, int, int *);	// Add two numbers

int main()
{
	int c;		// Final result will be stored here
	int *dev_c;	// Allocation of device memory done here.

	hipMalloc((void**)&dev_c, sizeof(int));

	add << <1, 1 >> > (23, 10, dev_c);

	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("Result: %d", c);
	hipFree(dev_c);
}

__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}
