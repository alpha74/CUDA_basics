#include "hip/hip_runtime.h"
// Using class members inside device code
// Author: alpha74

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

class Person
{
	private:
		int id;
		int age;
		char citizen_type;

	public:

		// Default ctor
		__host__ __device__ Person()
		{
			id = -1;
			age = -1;
			citizen_type = 'N';
		}

		// Copy ctor
		__host__ __device__ Person(const Person &P)
		{
			id = P.id;
			age = P.age;
			citizen_type = P.citizen_type;
		}

		// Setter
		__host__ __device__ void set(int s_id, int s_age )
		{
			id = s_id;
			age = s_age;
		}

		__host__ __device__ void set_ct(char c)
		{
			citizen_type = c;
		}

		// Display
		__host__ void display()
		{
			cout << "\n ID: " << id << "  Age: " << age << "  Citizen_Type: " << citizen_type;
		}

		// Getter
		__host__ __device__ int get_id()
		{
			return id;
		}

		__host__ __device__ int get_age()
		{
			return age;
		}

		__host__ __device__ char get_ct()
		{
			return citizen_type;
		}

		// Dtor
		__host__ __device__ ~Person()
		{
			;
		}

};

const int N = 100;

// Kernel declaration
__global__ void kernel_set_ct(Person *);
int min(int &, int &);

int main()
{
	Person Public[N];

	// Initialize age and id
	for (int i = 0; i < N; i++)
	{
		Public[i].set(i, min((i * 2 + 10), 89));
	}

	Person *dev_Public;

	// Allocate on device
	hipMalloc((void **)&dev_Public, N * sizeof(Person));
	// Copy contents
	hipMemcpy(dev_Public, Public, N * sizeof(Person), hipMemcpyHostToDevice);

	// Launch kernel
	kernel_set_ct << <5, 10 >> > (dev_Public);

	// Copy back
	hipMemcpy(Public, dev_Public, N * sizeof(Person), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	// Display
	cout << "\n Contents: ";

	for (int i = 0; i < N; i++)
	{
		Public[i].display();
	}

	// Deallocate memory
	hipFree(dev_Public);

	return 0;
}

// Kernel definition
__global__ void kernel_set_ct(Person *Public)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;

	// We can also send the limit as a passed argument and then use it, removing the dependency on variable N.
	while (tid < N)
	{
		// Set Citizen type using age:
		int age = Public[tid].get_age();
		char c = 'x';

		// Simple cases
		if (age >= 0 && age <= 12)
			c = 'K';	// Kid

		else if (age >= 13 && age <= 19)
			c = 'T';	// Teen

		else if (age >= 20 && age <= 35)
			c = 'A';	// Adult

		else if (age >= 36 && age <= 55)
			c = 'M';	// Middle

		else
			c = 'S';	// Senior


		// Set
		Public[tid].set_ct(c);

		// Increment tid
		tid += gridDim.x * blockDim.x ;
	}
}

int min(int &a, int &b)
{
	if (a < b)
		return a;
	else
		return b;
}
