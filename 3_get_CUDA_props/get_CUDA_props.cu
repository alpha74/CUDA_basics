// Getting information about CUDA Device
// Author: alpha74

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

int main()
{
	hipDeviceProp_t prop;

	int devcount;
	
	// Returns the number of CUDA devices attached to system
	hipGetDeviceCount(&devcount);

	// Iterate and fetch the details of each deviceID
	for (int i = 0; i < devcount; i++)
	{
		hipGetDeviceProperties(&prop, i);

		printf("\n\n Name: %s", prop.name);
		printf("\n Multiprocessor count: %d", prop.multiProcessorCount);
		printf("\n Clock rate: %d", prop.clockRate);
		printf("\n Compute Cap: %d.%d", prop.major, prop.minor);
	}
}
