// Getting information about CUDA

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

int main()
{
	hipDeviceProp_t prop;

	int devcount;

	hipGetDeviceCount(&devcount);

	for (int i = 0; i < devcount; i++)
	{
		hipGetDeviceProperties(&prop, i);

		printf("\n\n Name: %s", prop.name);
		printf("\n Multiprocessor count: %d", prop.multiProcessorCount);
		printf("\n Clock rate: %d", prop.clockRate);
		printf("\n Compute Cap: %d.%d", prop.major, prop.minor);
	}
}
